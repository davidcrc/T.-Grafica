#include <ctime>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define RED 2
#define GREEN 1
#define BLUE 0


using namespace std;

extern "C"
hipError_t cuda_main()
{
    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;


    int devCount;
    hipGetDeviceCount(&devCount);
    wcout << "Dispositivo CUDA : " << endl << endl;

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "Memoria global:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "Memoria compartida:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "Memoria constante: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "Registros por bloque: " << props.regsPerBlock << endl << endl;
        wcout << "Multi Processor Count: " << props.multiProcessorCount << endl << endl;

        wcout << "Tamano Warp :         " << props.warpSize << endl;
        wcout << "Threads por block: " << props.maxThreadsPerBlock << endl;
        wcout << "Dimension Max block : [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "Dimension Max grid :  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }

    return hipGetLastError();
}

//funcion device
__device__ unsigned char clamp(int value){
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return (unsigned char)value;
}

//funcion global de filtro
__global__ void Convolucion(char *M, unsigned char *imageInput, int width, int height, \
        unsigned int maskWidth,unsigned char *imageOutput){

    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0;

    int start_row = row - (maskWidth/2);
    int start_col = col - (maskWidth/2);

    for(int i = 0; i < maskWidth; i++){
        for(int j = 0; j < maskWidth; j++ ){
            if((start_col + j >=0 && start_col + j < width) \
                    &&(start_row + i >=0 && start_row + i < height)){
                Pvalue += imageInput[(start_row + i)*width+(start_col + j)] * M[i*maskWidth+j];
            }
        }
    }
    imageOutput[row*width+col] = clamp(Pvalue);
}

//funcion global de escala de grises
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 \
                                     + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}

//funcion global de filtro
__global__ void FilterColor(char *M, unsigned char *imageInput, int width, int height, \
    unsigned int maskWidth,unsigned char *imageOutput, float coef){

    unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
    unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

    int Pvalue = 0.0;

    int start_row = row - (maskWidth/2);
    int start_col = col - (maskWidth/2);

    for(int i = 0; i < maskWidth; i++){
        for(int j = 0; j < maskWidth; j++ ){
            if((start_col + j >=0 && start_col + j < width) \
                    &&(start_row + i >=0 && start_row + i < height)){
                Pvalue += imageInput[(start_row + i)*width+(start_col + j)] * M[i*maskWidth+j]*coef;
            }
        }
    }
    imageOutput[row*width+col] = clamp(Pvalue);

}
hipError_t  convolucionCuda(int width, int height, int size, int sizeGray, unsigned char *dataRawImage, unsigned char *&h_imageOutput, \
                             int MASK_WIDTH, char *h_M){
    hipError_t error = hipSuccess;
    clock_t startGPU, endGPU;
    double gpu_time;
    int sizeM = sizeof(char)*MASK_WIDTH*MASK_WIDTH;
    char *d_M;
    unsigned char *d_dataRawImage, *d_imageOutput, *d_sobelOutput;

    //Reserva de Memoria para M
    error = hipMalloc((void**)&d_M,sizeM);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_M\n");
        exit(-1);
    }

    //Reserva de Memoria para d_dataRawImage
    error = hipMalloc((void**)&d_dataRawImage,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_dataRawImage\n");
        exit(-1);
    }

    //Reserva de Memoria para d_imageOutput
    error = hipMalloc((void**)&d_imageOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageOutput\n");
        exit(-1);
    }

    //Reserva de Memoria para d_sobelOutput
    error = hipMalloc((void**)&d_sobelOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_sobelOutput\n");
        exit(-1);
    }

    //Algoritmo Paralelo con CUDA
    startGPU = clock();

    error = hipMemcpy(d_dataRawImage,dataRawImage,size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
       printf("Error copiando los datos de dataRawImage a d_dataRawImage \n");
       exit(-1);
    }

    error = hipMemcpy(d_M,h_M,sizeM, hipMemcpyHostToDevice);
    if(error != hipSuccess){
       printf("Error copiando los datos de h_M a d_M \n");
       exit(-1);
    }

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    //llamamos la funcion de escala de grises
    img2gray<<<dimGrid,dimBlock>>>(d_dataRawImage,width,height,d_imageOutput);
    //Sincronizamos
    hipDeviceSynchronize();
    //llamamos la funcion de filtro de sobel
    Convolucion<<<dimGrid,dimBlock>>>(d_M, d_imageOutput,width,height,MASK_WIDTH,d_sobelOutput);
    hipMemcpy(h_imageOutput,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);

    endGPU = clock();
    //fin algoritmo Paralelo con CUDA

    //imprimir tiempos de ejecucion
    gpu_time = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Paralelo CUDA: %.10f\n",gpu_time);

    //limpiar memoria
    hipFree(d_dataRawImage);
    hipFree(d_imageOutput);
    hipFree(d_M);
    hipFree(d_sobelOutput);

    return hipGetLastError();
}


hipError_t  convolucionCudaColor(int width, int height, int sizeGray, unsigned char * splB, unsigned char * splG, unsigned char * splR,\
                    unsigned char *&h_imageOutputB,unsigned char *&h_imageOutputG,unsigned char *&h_imageOutputR,\
                    int MASK_WIDTH, char *h_M, float coef){

    hipError_t error = hipSuccess;
    clock_t startGPU, endGPU;
    double gpu_time;
    int sizeM = sizeof(char)*MASK_WIDTH*MASK_WIDTH;
    char *d_M;
    unsigned char *d_dataRawImageB, *d_dataRawImageG, *d_dataRawImageR, *d_sobelOutput;

    //Reserva de Memoria para M
    error = hipMalloc((void**)&d_M,sizeM);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_M\n");
        exit(-1);
    }

    error = hipMalloc((void**)&d_dataRawImageB,sizeGray);
    error = hipMalloc((void**)&d_dataRawImageG,sizeGray);
    error = hipMalloc((void**)&d_dataRawImageR,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_dataRawImage\n");
        exit(-1);
    }


    //Reserva de Memoria para d_sobelOutput
    error = hipMalloc((void**)&d_sobelOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_sobelOutput\n");
        exit(-1);
    }

    //Algoritmo Paralelo con CUDA
    startGPU = clock();


    error = hipMemcpy(d_M,h_M,sizeM, hipMemcpyHostToDevice);
    if(error != hipSuccess){
       printf("Error copiando los datos de h_M a d_M  \n");
       exit(-1);
    }

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);

    hipDeviceSynchronize();

    error = hipMemcpy(d_dataRawImageB,splB,sizeGray, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de splB a d_dataRawImage \n");
        exit(-1);
    }
    FilterColor<<<dimGrid,dimBlock>>>(d_M, d_dataRawImageB,width,height,MASK_WIDTH,d_sobelOutput,coef);
    hipMemcpy(h_imageOutputB,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    ///////////////////////////

    error = hipMemcpy(d_dataRawImageG,splG,sizeGray, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de splG a d_dataRawImage \n");
        exit(-1);
    }


    FilterColor<<<dimGrid,dimBlock>>>(d_M,d_dataRawImageG,width,height,MASK_WIDTH,d_sobelOutput,coef);
    hipMemcpy(h_imageOutputG,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    /////////////////////////////

    error = hipMemcpy(d_dataRawImageR,splR,sizeGray, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de splR a d_dataRawImage \n");
        exit(-1);
    }

    FilterColor<<<dimGrid,dimBlock>>>(d_M,d_dataRawImageR,width,height,MASK_WIDTH,d_sobelOutput,coef);
    hipMemcpy(h_imageOutputR,d_sobelOutput,sizeGray,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    endGPU = clock();
    //fin algoritmo Paralelo con CUDA

    //imprimir tiempos de ejecucion
    gpu_time = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Paralelo CUDA: %.10f\n",gpu_time);

    //limpiar memoria
    hipFree(d_dataRawImageB);
    hipFree(d_dataRawImageG);
    hipFree(d_dataRawImageR);

    hipFree(d_M);
    hipFree(d_sobelOutput);

    return hipGetLastError();
}
